#include "hip/hip_runtime.h"
#include <cli_parser.hpp>
#include <cpumultiply.hpp>  //! header file for tiling
#include <tiling.hpp>       //! header file for tiling
#include <coloring.cuh>

#include <stdio.h>
#include <iostream>
#include <numeric>
// #include <thrust/host_vector.h>
// #include <thrust/device_vector.h>
#include <asc.cuh>

#include <defines.hpp>
#include <kernel_setup.hpp>
#include <coloringCounters.cuh>

#include <cpu_coloring.hpp>

void printResult(const apa22_coloring::Counters& sum,
                 const apa22_coloring::Counters& max) {
    printf("Total Collisions\n");
    const auto start_bw = apa22_coloring::start_bit_width;
    for (uint i = 0; i < apa22_coloring::num_bit_widths; ++i) {
      printf("Mask width: %d, Collisions: %d\n", i+start_bw, sum.m[i]);
    }

    printf("Max Collisions per Node\n");
    for (uint i = 0; i < apa22_coloring::num_bit_widths; ++i) {
      printf("Mask width: %d, Collisions: %d\n", i+start_bw, max.m[i]);
    }
}

int main(int argc, char const *argv[]) {
  using namespace apa22_coloring;

  int mat_nr = 2;          //Default value
  chCommandLineGet<int>(&mat_nr, "m", argc, argv);

  const char* inputMat = def::choseMat(mat_nr);

  int* row_ptr;
  int* col_ptr;
  double* val_ptr;  // create pointers for matrix in csr format
  int m_rows;
  // const int m_rows =
  //     cpumultiplyDloadMTX(inputMat, &row_ptr, &col_ptr, &val_ptr);

  int* ndc_;     // array with indices of each tile in all slices
  // int* slices_;  // array with nodes grouped in slices
  // int* offsets_;
  // simple_tiling(m_rows, number_of_tiles, row_ptr, col_ptr, &slices_, &ndc_,
  //               &offsets_);
  // cpumultiplyDpermuteMatrix(number_of_tiles, 1, ndc_, slices_, row_ptr, col_ptr,
  //                           val_ptr, &row_ptr, &col_ptr, &val_ptr, true);
  int number_of_tiles;
  int shMem_size_bytes;
  kernel_setup<true>(inputMat, row_ptr, col_ptr, val_ptr, ndc_, m_rows, number_of_tiles, shMem_size_bytes, 300);
  printf("Nr_tiles: %d\n", number_of_tiles);
  printf("shMem: %d\n", shMem_size_bytes);
  printf("M-row %d", m_rows);
  std::cout << std::endl;

  int* d_row_ptr;
  int* d_col_ptr;
  int* d_tile_boundaries;
  
  size_t row_ptr_len = m_rows + 1;
  size_t col_ptr_len = row_ptr[m_rows];
  size_t tile_bound_len = number_of_tiles + 1;

  printf("Post hipMalloc");
  std::cout << std::endl;

  //==================================================
  // Allocate memory for partitioned matrix on device
  //==================================================
  hipMalloc((void**)&d_row_ptr, row_ptr_len * sizeof(int));
  hipMalloc((void**)&d_col_ptr, col_ptr_len * sizeof(int));
  hipMalloc((void**)&d_tile_boundaries, tile_bound_len * sizeof(int));


  hipMemcpy(d_row_ptr, row_ptr, row_ptr_len * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(d_col_ptr, col_ptr, col_ptr_len * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(d_tile_boundaries, ndc_, tile_bound_len * sizeof(int),
             hipMemcpyHostToDevice);

  //==========================================================
  // Allocate memory for intermediate block reduction results
  //==========================================================
  // For each bit_width we allocate a counter for each block and for each hash function
  SOACounters h_soa_total1;
  for (int i = 0; i < hash_params.len; ++i) {
    hipMalloc((void**)&(h_soa_total1.m[i]), num_bit_widths * number_of_tiles * sizeof(int));
  }
  SOACounters h_soa_max1;
  for (int i = 0; i < hash_params.len; ++i) {
    hipMalloc((void**)&(h_soa_max1.m[i]), num_bit_widths * number_of_tiles * sizeof(int));
  }
  SOACounters h_soa_total2;
  for (int i = 0; i < hash_params.len; ++i) {
    hipMalloc((void**)&(h_soa_total2.m[i]), num_bit_widths * number_of_tiles * sizeof(int));
  }
  SOACounters h_soa_max2;
  for (int i = 0; i < hash_params.len; ++i) {
    hipMalloc((void**)&(h_soa_max2.m[i]), num_bit_widths * number_of_tiles * sizeof(int));
  }
  //==========================================================================
  // Allocate memory for structs holding pointers to intermediate result alloc
  //==========================================================================
  SOACounters* d_soa_total1;
  hipMalloc((void**)&d_soa_total1, sizeof(SOACounters));
  SOACounters* d_soa_max1;
  hipMalloc((void**)&d_soa_max1, sizeof(SOACounters));
  SOACounters* d_soa_total2;
  hipMalloc((void**)&d_soa_total2, sizeof(SOACounters));
  SOACounters* d_soa_max2;
  hipMalloc((void**)&d_soa_max2, sizeof(SOACounters));

  hipMemcpy(d_soa_total1, &h_soa_total1, sizeof(SOACounters),
             hipMemcpyHostToDevice);
  hipMemcpy(d_soa_max1, &h_soa_max1, sizeof(SOACounters),
             hipMemcpyHostToDevice);
  hipMemcpy(d_soa_total2, &h_soa_total2, sizeof(SOACounters),
             hipMemcpyHostToDevice);
  hipMemcpy(d_soa_max2, &h_soa_max2, sizeof(SOACounters),
             hipMemcpyHostToDevice);

  //==================================
  // Allocate memory for return values
  //==================================
  Counters* d_total1;
  hipMalloc((void**)&d_total1, hash_params.len * sizeof(Counters));
  Counters* d_max1;
  hipMalloc((void**)&d_max1, hash_params.len * sizeof(Counters));
  Counters* d_total2;
  hipMalloc((void**)&d_total2, hash_params.len * sizeof(Counters));
  Counters* d_max2;
  hipMalloc((void**)&d_max2, hash_params.len * sizeof(Counters));
  
  // calc shMem
  size_t shMem_bytes = shMem_size_bytes;
  dim3 gridSize(number_of_tiles);
  dim3 blockSize(THREADS);

  int max_nodes, max_edges;
  get_MaxTileSize(number_of_tiles, ndc_, row_ptr, &max_nodes, &max_edges);
  int Necessary_shMem = 2 * (max_nodes + max_edges + 1) * sizeof(int);
  printf("max_nodes: %d \n", max_nodes);
  printf("max_nodes: %d \n", max_edges);
  printf("kernel_setup: %d \n", shMem_bytes);
  printf("Necessary_shMem: %d \n", Necessary_shMem);
  printf("Pre Kernel");
  std::cout << std::endl;

  // hipFuncSetAttribute(reinterpret_cast<const void*>(coloring1Kernel<int>), hipFuncAttributeMaxDynamicSharedMemorySize, 98304);
  // coloring1Kernel<<<gridSize, blockSize, shMem_bytes>>>(
  //     d_row_ptr, d_col_ptr, d_tile_boundaries,
  //     max_nodes, max_edges, d_soa_total1, d_soa_max1, d_total1, d_max1);
  coloring2Kernel<<<gridSize, blockSize, shMem_bytes>>>(
      d_row_ptr, d_col_ptr, d_tile_boundaries, max_nodes, max_edges,
      d_soa_total1, d_soa_max1, d_soa_total2, d_soa_max2,
      d_total1, d_max1, d_total2, d_max2);
  hipDeviceSynchronize();

  printf("Post Kernel");
  std::cout << std::endl;

  //====================================
  // Allocate memory for results on host
  //====================================
  std::unique_ptr<Counters[]> total1(new Counters[hash_params.len]);
  std::unique_ptr<Counters[]> max1(new Counters[hash_params.len]);
  std::unique_ptr<Counters[]> total2(new Counters[hash_params.len]);
  std::unique_ptr<Counters[]> max2(new Counters[hash_params.len]);
  hipMemcpy(total1.get(), d_total1, hash_params.len * sizeof(Counters),
             hipMemcpyDeviceToHost);
  hipMemcpy(max1.get(), d_max1, hash_params.len * sizeof(Counters),
             hipMemcpyDeviceToHost);
  hipMemcpy(total2.get(), d_total2, hash_params.len * sizeof(Counters),
             hipMemcpyDeviceToHost);
  hipMemcpy(max2.get(), d_max2, hash_params.len * sizeof(Counters),
             hipMemcpyDeviceToHost);

  printResult(total2[0], max2[0]);

  auto redBinaryOp = [](auto lhs, auto rhs){return rhs > lhs ? rhs : lhs;};
  auto transBinaryOp = [](auto lhs, auto rhs){return rhs - lhs;};
  int max_node_degree = std::transform_reduce(row_ptr,
                                              row_ptr + m_rows,
                                              row_ptr + 1,
                                              0,
                                              redBinaryOp,
                                              transBinaryOp);
  Counters cpu_max, cpu_total;
  cpuDist2(row_ptr, col_ptr, m_rows, max_node_degree, &cpu_total, &cpu_max);
  
  printf("CPU results");
  printResult(cpu_total, cpu_max);

  //=================
  // Free device mem
  //=================
  hipFree(d_total1);
  hipFree(d_max1);
  hipFree(d_total2);
  hipFree(d_max2);

  for (int i = 0; i < hash_params.len; ++i) {
    hipFree(h_soa_total1.m[i]);
    hipFree(h_soa_max1.m[i]);
    hipFree(h_soa_total2.m[i]);
    hipFree(h_soa_max2.m[i]);
  }
  hipFree(d_soa_total1);
  hipFree(d_soa_max1);
  hipFree(d_soa_total2);
  hipFree(d_soa_max2);

  hipFree(d_row_ptr);
  hipFree(d_col_ptr);
  hipFree(d_tile_boundaries);

  printf("Post hipFree");
  std::cout << std::endl;

  // thrust::host_vector<int> row(row_ptr, row_ptr + m_rows + 1);
  // thrust::host_vector<int> col(col_ptr, col_ptr + row_ptr[m_rows]);
  // thrust::host_vector<double> nnz(val_ptr, val_ptr + row_ptr[m_rows]);
  
  // thrust::device_vector<int> d_row = row;
  // thrust::device_vector<int> d_col = col;
  // thrust::device_vector<double> d_nnz = nnz;

  // namespace asc18 = asc_hash_graph_coloring;
  // asc18::cusparse_distance1(d_nnz, d_row, d_col, 1);

  // delete total;
  // delete max;
	delete[] row_ptr;
	delete[] col_ptr;
	delete[] val_ptr;
	delete[] ndc_;
  return 0;
}