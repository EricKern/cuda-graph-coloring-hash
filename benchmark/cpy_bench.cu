#include <nvbench/nvbench.cuh>
#include <hipsparse.h>
#include <thrust/device_vector.h>

#include "coloring.cuh"
#include "defines.hpp"
#include "coloring_counters.cuh"
#include "copy_kernel.cuh"
#include "setup.cuh"
#include "mat_loader.hpp"
#include "cli_bench.cu"
#include "bench_util.cuh"

using BLOCKS_SM = nvbench::enum_type_list<1, 2, 4>;
using THREADS_SM = nvbench::enum_type_list<512, 640, 768, 896, 1024>;
using THREADS_SM_Fine = nvbench::enum_type_list<256, 384, 512, 640, 768, 896, 1024>;
static constexpr const char* Blocks_SM_key = "BLK_SM";
static constexpr const char* Threads_SM_key = "THREADS_SM";
static constexpr const char* SM_ShMem_key = "Smem_SM";
std::vector<nvbench::int64_t> SM_ShMem_range = {32*1024, 64*1024};

using namespace apa22_coloring;

template <int MAX_THREADS_SM, int BLK_SM>
void Distance1(nvbench::state& state,
               nvbench::type_list<nvbench::enum_type<MAX_THREADS_SM>,
                                  nvbench::enum_type<BLK_SM>>) {
  constexpr int THREADS = MAX_THREADS_SM / BLK_SM;
  auto kernel = coloring1Kernel<THREADS, BLK_SM, int>;

  MatLoader& mat_loader = MatLoader::getInstance();
  Tiling tiling(D1, BLK_SM, mat_loader.row_ptr, mat_loader.m_rows,
                reinterpret_cast<void*>(kernel),
                state.get_int64(SM_ShMem_key));
  GPUSetupD1 gpu_setup(mat_loader.row_ptr, mat_loader.col_ptr,
                       tiling.tile_boundaries.get(), tiling.n_tiles);

  size_t shMem_bytes = tiling.tile_target_mem;
  dim3 gridSize(tiling.n_tiles);
  dim3 blockSize(THREADS);

  add_MatInfo(state);
  add_IOInfo(state, gridSize.x);

  state.exec(
      [&](nvbench::launch& launch) {
        kernel<<<gridSize, blockSize, shMem_bytes, launch.get_stream()>>>(
            gpu_setup.d_row_ptr,
            gpu_setup.d_col_ptr,
            gpu_setup.d_tile_boundaries,
            gpu_setup.blocks_total1,
            gpu_setup.blocks_max1,
            gpu_setup.d_total1,
            gpu_setup.d_max1);
      });
}

template <int MAX_THREADS_SM, int BLK_SM>
void Distance1Copy(nvbench::state& state,
                   nvbench::type_list<nvbench::enum_type<MAX_THREADS_SM>,
                                      nvbench::enum_type<BLK_SM>>) {
  constexpr int THREADS = MAX_THREADS_SM / BLK_SM;
  auto kernel = copyKernelDist1<THREADS, BLK_SM, int>;

  MatLoader& mat_loader = MatLoader::getInstance();
  Tiling tiling(D1, BLK_SM, mat_loader.row_ptr, mat_loader.m_rows,
                reinterpret_cast<void*>(kernel),
                state.get_int64(SM_ShMem_key));
  GPUSetupD1 gpu_setup(mat_loader.row_ptr, mat_loader.col_ptr,
                       tiling.tile_boundaries.get(), tiling.n_tiles);

  size_t shMem_bytes = tiling.tile_target_mem;
  dim3 gridSize(tiling.n_tiles);
  dim3 blockSize(THREADS);

  add_MatInfo(state);
  add_IOInfo(state, gridSize.x);

  state.exec(
      [&](nvbench::launch& launch) {
        kernel<<<gridSize, blockSize, shMem_bytes, launch.get_stream()>>>(
            gpu_setup.d_row_ptr,
            gpu_setup.d_col_ptr,
            gpu_setup.d_tile_boundaries,
            gpu_setup.blocks_total1,
            gpu_setup.blocks_max1,
            gpu_setup.d_total1,
            gpu_setup.d_max1);
      });
}

template <int MAX_THREADS_SM, int BLK_SM>
void Distance1CopyFence(nvbench::state& state,
                        nvbench::type_list<nvbench::enum_type<MAX_THREADS_SM>,
                                           nvbench::enum_type<BLK_SM>>) {
  constexpr int THREADS = MAX_THREADS_SM / BLK_SM;
  auto kernel = copyKernelD1TFence<THREADS, BLK_SM, int>;

  MatLoader& mat_loader = MatLoader::getInstance();
  Tiling tiling(D1, BLK_SM, mat_loader.row_ptr, mat_loader.m_rows,
                reinterpret_cast<void*>(kernel),
                state.get_int64(SM_ShMem_key));
  GPUSetupD1 gpu_setup(mat_loader.row_ptr, mat_loader.col_ptr,
                       tiling.tile_boundaries.get(), tiling.n_tiles);

  size_t shMem_bytes = tiling.tile_target_mem;
  dim3 gridSize(tiling.n_tiles);
  dim3 blockSize(THREADS);

  add_MatInfo(state);
  add_IOInfo(state, gridSize.x);

  state.exec(
      [&](nvbench::launch& launch) {
        kernel<<<gridSize, blockSize, shMem_bytes, launch.get_stream()>>>(
            gpu_setup.d_row_ptr,
            gpu_setup.d_col_ptr,
            gpu_setup.d_tile_boundaries,
            gpu_setup.blocks_total1,
            gpu_setup.blocks_max1,
            gpu_setup.d_total1,
            gpu_setup.d_max1);
      });
}

template <int MAX_THREADS_SM, int BLK_SM>
void Distance2(nvbench::state& state,
               nvbench::type_list<nvbench::enum_type<MAX_THREADS_SM>,
                                  nvbench::enum_type<BLK_SM>>) {
  constexpr int THREADS = MAX_THREADS_SM / BLK_SM;
  auto kernel = coloring2Kernel<THREADS, BLK_SM, int>;

  MatLoader& mat_loader = MatLoader::getInstance();
  Tiling tiling(D2, BLK_SM, mat_loader.row_ptr, mat_loader.m_rows,
                reinterpret_cast<void*>(kernel),
                state.get_int64(SM_ShMem_key));
  GPUSetupD2 gpu_setup(mat_loader.row_ptr, mat_loader.col_ptr,
                       tiling.tile_boundaries.get(), tiling.n_tiles);

  size_t shMem_bytes = tiling.tile_target_mem;
  dim3 gridSize(tiling.n_tiles);
  dim3 blockSize(THREADS);

  add_MatInfo(state);
  add_IOInfo(state, gridSize.x);

  state.exec(
      [&](nvbench::launch& launch) {
        kernel<<<gridSize, blockSize, shMem_bytes, launch.get_stream()>>>(
                gpu_setup.d_row_ptr,
                gpu_setup.d_col_ptr,
                gpu_setup.d_tile_boundaries,
                gpu_setup.blocks_total1,
                gpu_setup.blocks_max1,
                gpu_setup.blocks_total2,
                gpu_setup.blocks_max2,
                gpu_setup.d_total1,
                gpu_setup.d_max1,
                gpu_setup.d_total2,
                gpu_setup.d_max2);
      });
}

template <int MAX_THREADS_SM, int BLK_SM>
void Distance2Bank(nvbench::state& state,
                   nvbench::type_list<nvbench::enum_type<MAX_THREADS_SM>,
                                      nvbench::enum_type<BLK_SM>>) {
  constexpr int THREADS = MAX_THREADS_SM / BLK_SM;
  auto kernel = coloring2KernelBank<THREADS, BLK_SM, int>;

  MatLoader& mat_loader = MatLoader::getInstance();
  Tiling tiling(D2_SortNet, BLK_SM, mat_loader.row_ptr, mat_loader.m_rows,
                reinterpret_cast<void*>(kernel),
                state.get_int64(SM_ShMem_key));
  GPUSetupD2 gpu_setup(mat_loader.row_ptr, mat_loader.col_ptr,
                       tiling.tile_boundaries.get(), tiling.n_tiles);

  size_t shMem_bytes = tiling.tile_target_mem;
  dim3 gridSize(tiling.n_tiles);
  dim3 blockSize(THREADS);

  add_MatInfo(state);
  add_IOInfo(state, gridSize.x);

  state.exec([&](nvbench::launch& launch) {
    kernel<<<gridSize, blockSize, shMem_bytes, launch.get_stream()>>>(
        gpu_setup.d_row_ptr, gpu_setup.d_col_ptr, gpu_setup.d_tile_boundaries,
        tiling.max_node_degree, gpu_setup.blocks_total1, gpu_setup.blocks_max1,
        gpu_setup.blocks_total2, gpu_setup.blocks_max2, gpu_setup.d_total1,
        gpu_setup.d_max1, gpu_setup.d_total2, gpu_setup.d_max2);
  });
}

template <int MAX_THREADS_SM, int BLK_SM>
void Distance2Copy(nvbench::state& state,
                   nvbench::type_list<nvbench::enum_type<MAX_THREADS_SM>,
                                      nvbench::enum_type<BLK_SM>>) {
  constexpr int THREADS = MAX_THREADS_SM / BLK_SM;
  auto kernel = copyKernelDist2<THREADS, BLK_SM, int>;

  MatLoader& mat_loader = MatLoader::getInstance();
  Tiling tiling(D2, BLK_SM, mat_loader.row_ptr, mat_loader.m_rows,
                reinterpret_cast<void*>(kernel),
                state.get_int64(SM_ShMem_key));
  GPUSetupD2 gpu_setup(mat_loader.row_ptr, mat_loader.col_ptr,
                       tiling.tile_boundaries.get(), tiling.n_tiles);

  size_t shMem_bytes = tiling.tile_target_mem;
  dim3 gridSize(tiling.n_tiles);
  dim3 blockSize(THREADS);

  add_MatInfo(state);
  add_IOInfo(state, gridSize.x);

  state.exec(
      [&](nvbench::launch& launch) {
        kernel<<<gridSize, blockSize, shMem_bytes, launch.get_stream()>>>(
                gpu_setup.d_row_ptr,
                gpu_setup.d_col_ptr,
                gpu_setup.d_tile_boundaries,
                gpu_setup.blocks_total1,
                gpu_setup.blocks_max1,
                gpu_setup.blocks_total2,
                gpu_setup.blocks_max2,
                gpu_setup.d_total1,
                gpu_setup.d_max1,
                gpu_setup.d_total2,
                gpu_setup.d_max2);
      });
}

void Distance1cusparse(nvbench::state &state) {
    hipsparseHandle_t handle;
	hipsparseCreate(&handle);
	hipsparseMatDescr_t descG;
	// creates descriptor for 0-based indexing and general matrix by default
	hipsparseCreateMatDescr(&descG);

	hipsparseColorInfo_t info;
	hipsparseCreateColorInfo(&info);

	// fraction of vertices that has to be colored iteratively before falling back
	// to giving every leftover node an unique color
	constexpr double fraction = 1.0;

	int num_colors; // will be updated by cusparse

    MatLoader& mat_loader = MatLoader::getInstance();

    double* d_val_ptr;
    int* d_col_ptr;
    int* d_row_ptr; 
    hipMalloc((void**)&d_val_ptr, mat_loader.row_ptr[mat_loader.m_rows] * sizeof(double));
    hipMalloc((void**)&d_col_ptr, mat_loader.row_ptr[mat_loader.m_rows] * sizeof(int));
    hipMalloc((void**)&d_row_ptr, (mat_loader.m_rows + 1) * sizeof(int));
    
    hipMemcpy(d_val_ptr, mat_loader.val_ptr, mat_loader.row_ptr[mat_loader.m_rows] * sizeof(double),
                hipMemcpyHostToDevice);
    hipMemcpy(d_col_ptr, mat_loader.col_ptr, mat_loader.row_ptr[mat_loader.m_rows] * sizeof(int),
                hipMemcpyHostToDevice);
    hipMemcpy(d_row_ptr, mat_loader.row_ptr, mat_loader.m_rows * sizeof(int),
                hipMemcpyHostToDevice);

    thrust::device_vector<int> coloring(mat_loader.row_ptr[mat_loader.m_rows]);

    add_MatInfo(state);
    size_t in_elem = mat_loader.m_rows + mat_loader.row_ptr[mat_loader.m_rows];
    state.add_global_memory_reads<int>(in_elem);

    state.exec(nvbench::exec_tag::sync, [&](nvbench::launch& launch) {		
		    hipsparseDcsrcolor(handle,
		                      mat_loader.m_rows,
		                      mat_loader.row_ptr[mat_loader.m_rows],
		                      descG,
		                      d_val_ptr,
		                      d_row_ptr,
		                      d_col_ptr,
		                      &fraction,
		                      &num_colors,
		                      thrust::raw_pointer_cast(coloring.data()),
		                      nullptr, // don't need reordering
		                      info);
    });

    state.add_element_count(num_colors, "Colors");
	hipsparseDestroyColorInfo(info);
	hipsparseDestroyMatDescr(descG);
	hipsparseDestroy(handle);
    hipFree(d_val_ptr);
    hipFree(d_col_ptr);
    hipFree(d_row_ptr);
}


NVBENCH_BENCH_TYPES(Distance1, NVBENCH_TYPE_AXES(THREADS_SM_Fine, BLOCKS_SM))
    .set_type_axes_names({Threads_SM_key, Blocks_SM_key})
    .add_int64_axis(SM_ShMem_key, SM_ShMem_range);

// NVBENCH_BENCH_TYPES(Distance1Copy, NVBENCH_TYPE_AXES(THREADS_SM, BLOCKS_SM))
//     .set_type_axes_names({Threads_SM_key, Blocks_SM_key})
//     .add_int64_axis(SM_ShMem_key, SM_ShMem_range);

// NVBENCH_BENCH_TYPES(Distance1CopyFence, NVBENCH_TYPE_AXES(THREADS_SM, BLOCKS_SM))
//     .set_type_axes_names({Threads_SM_key, Blocks_SM_key})
//     .add_int64_axis(SM_ShMem_key, SM_ShMem_range);

// NVBENCH_BENCH_TYPES(Distance2, NVBENCH_TYPE_AXES(THREADS_SM, BLOCKS_SM))
//     .set_type_axes_names({Threads_SM_key, Blocks_SM_key})
//     .add_int64_axis(SM_ShMem_key, SM_ShMem_range);

// NVBENCH_BENCH_TYPES(Distance2Bank, NVBENCH_TYPE_AXES(THREADS_SM, BLOCKS_SM))
//     .set_type_axes_names({Threads_SM_key, Blocks_SM_key})
//     .add_int64_axis(SM_ShMem_key, SM_ShMem_range);

// NVBENCH_BENCH_TYPES(Distance2Copy, NVBENCH_TYPE_AXES(THREADS_SM, BLOCKS_SM))
//     .set_type_axes_names({Threads_SM_key, Blocks_SM_key})
//     .add_int64_axis(SM_ShMem_key, SM_ShMem_range);
// NVBENCH_BENCH(Distance1cusparse);
