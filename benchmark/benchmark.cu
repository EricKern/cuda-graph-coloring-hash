#include <nvbench/nvbench.cuh>

#include "kernel_setup.hpp"
#include "defines.hpp"
#include "coloringCounters.cuh"
#include "copyKernel.cuh"

using namespace apa22_coloring;

void initBenchmark(const char* Matrix, int* &d_row_ptr, int* &d_col_ptr, int* &d_tile_boundaries,
                    SOACounters* &d_soa_total, SOACounters* &d_soa_max, Counters* &d_total,
                    Counters* &d_max, int &max_nodes, int &max_edges, int &shMem_size_bytes,
                    int &number_of_tiles, size_t &size){
    int* row_ptr;
    int* col_ptr;
    double* val_ptr;
    int m_rows;
    int* ndc_;
    kernel_setup(Matrix, row_ptr, col_ptr, val_ptr, ndc_, m_rows, number_of_tiles, shMem_size_bytes, 300);
  
    size_t row_ptr_len = m_rows + 1;
    size_t col_ptr_len = size = row_ptr[m_rows];
    size_t tile_bound_len = number_of_tiles + 1;

    hipMalloc((void**)&d_row_ptr, row_ptr_len * sizeof(int));
    hipMalloc((void**)&d_col_ptr, col_ptr_len * sizeof(int));
    hipMalloc((void**)&d_tile_boundaries, tile_bound_len * sizeof(int));


    hipMemcpy(d_row_ptr, row_ptr, row_ptr_len * sizeof(int),
             hipMemcpyHostToDevice);
    hipMemcpy(d_col_ptr, col_ptr, col_ptr_len * sizeof(int),
             hipMemcpyHostToDevice);
    hipMemcpy(d_tile_boundaries, ndc_, tile_bound_len * sizeof(int),
             hipMemcpyHostToDevice);

    // For each bit_width we allocate a counter for each block and for each hash function
    SOACounters h_soa_total;
    for (int i = 0; i < hash_params.len; ++i) {
        hipMalloc((void**)&(h_soa_total.m[i]), num_bit_widths * number_of_tiles * sizeof(int));
    }
    SOACounters h_soa_max;
    for (int i = 0; i < hash_params.len; ++i) {
        hipMalloc((void**)&(h_soa_max.m[i]), num_bit_widths * number_of_tiles * sizeof(int));
    }
   
    hipMalloc((void**)&d_soa_total, sizeof(SOACounters));

    hipMalloc((void**)&d_soa_max, sizeof(SOACounters));

    hipMemcpy(d_soa_total, &h_soa_total, sizeof(SOACounters),
             hipMemcpyHostToDevice);
    hipMemcpy(d_soa_max, &h_soa_max, sizeof(SOACounters),
             hipMemcpyHostToDevice);

 
    hipMalloc((void**)&d_total, hash_params.len * sizeof(Counters));
    hipMalloc((void**)&d_max, hash_params.len * sizeof(Counters));
  
    get_MaxTileSize(number_of_tiles, ndc_, row_ptr, &max_nodes, &max_edges);

    delete[] row_ptr;
    delete[] col_ptr;
	delete[] val_ptr;
	delete[] ndc_;
}

void copyBench(nvbench::state &state){
    int* d_row_ptr;
    int* d_col_ptr;
    int* d_tile_boundaries;
    SOACounters* d_soa_total;
    SOACounters* d_soa_max;
    Counters* d_total;
    Counters* d_max;
    int shMem_size_bytes;
    int number_of_tiles;
    int max_nodes;
    int max_edges;
    size_t size;

    state.set_timeout(-1);
    initBenchmark(def::Mat3, d_row_ptr, d_col_ptr, d_tile_boundaries, d_soa_total, d_soa_max,
                d_total, d_max, max_nodes, max_edges, shMem_size_bytes, number_of_tiles, size);
    
    state.add_element_count(size, "Elements");
    state.add_global_memory_reads<int>(size);
    //state.add_global_memory_writes<Counters>(size);
    state.collect_dram_throughput();
    state.collect_l1_hit_rates();
    state.collect_l2_hit_rates();
    state.collect_loads_efficiency();
    state.collect_stores_efficiency();

    dim3 gridSize(number_of_tiles);
    dim3 blockSize(THREADS);

    state.exec([&](nvbench::launch &launch){
        copyKernel<<<gridSize, blockSize, shMem_size_bytes, launch.get_stream()>>>(
            d_row_ptr, d_col_ptr, d_tile_boundaries, max_nodes, max_edges, d_soa_total,
            d_soa_max, d_total, d_max);
    });

    hipFree(d_row_ptr);
    hipFree(d_col_ptr);
    hipFree(d_tile_boundaries);
    hipFree(d_soa_total);
    hipFree(d_soa_max);
    hipFree(d_total);
    hipFree(d_max);
}

void coloring1Bench(nvbench::state &state){
    int* d_row_ptr;
    int* d_col_ptr;
    int* d_tile_boundaries;
    SOACounters* d_soa_total;
    SOACounters* d_soa_max;
    Counters* d_total;
    Counters* d_max;
    int shMem_size_bytes;
    int number_of_tiles;
    int max_nodes;
    int max_edges;
    size_t size;

    state.set_timeout(-1);
    initBenchmark(def::Mat3, d_row_ptr, d_col_ptr, d_tile_boundaries, d_soa_total, d_soa_max,
                d_total, d_max, max_nodes, max_edges, shMem_size_bytes, number_of_tiles, size);
    
    state.add_element_count(size, "Elements");
    state.add_global_memory_reads<int>(size);
    //state.add_global_memory_writes<Counters>(size);
    state.collect_dram_throughput();
    state.collect_l1_hit_rates();
    state.collect_l2_hit_rates();
    state.collect_loads_efficiency();
    state.collect_stores_efficiency();

    dim3 gridSize(number_of_tiles);
    dim3 blockSize(THREADS);

    state.exec([&](nvbench::launch &launch){
        coloring1Kernel<<<gridSize, blockSize, shMem_size_bytes, launch.get_stream()>>>(
            d_row_ptr, d_col_ptr, d_tile_boundaries, max_nodes, max_edges, d_soa_total,
            d_soa_max, d_total, d_max);
    });

    hipFree(d_row_ptr);
    hipFree(d_col_ptr);
    hipFree(d_tile_boundaries);
    hipFree(d_soa_total);
    hipFree(d_soa_max);
    hipFree(d_total);
    hipFree(d_max);
}


NVBENCH_BENCH(copyBench);
NVBENCH_BENCH(coloring1Bench);