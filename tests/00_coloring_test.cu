#include "gtest/gtest.h"


#include <cpumultiply.hpp>  //! header file for tiling
#include <tiling.hpp>       //! header file for tiling
#include <defines.hpp>

#include <00_Reduce_inl.cuh>
#include <numeric>
#include <00_Partition2ShMem.cuh>

#include <cpu_coloring.hpp>

namespace {
// To use a test fixture, derive a class from testing::Test.
class ColoringEnv : public testing::Test {
 protected:  // You should make the members protected s.t. they can be
             // accessed from sub-classes.
  // virtual void SetUp() will be called before each test is run.  You
  // should define it if you need to initialize the variables.
  // Otherwise, this can be skipped.
  void SetUp() override {
    const char* inputMat = def::Mat2;
    number_of_tiles = 24;

    m_rows = cpumultiplyDloadMTX(inputMat, &row_ptr, &col_ptr, &val_ptr);

    simple_tiling(m_rows, number_of_tiles, row_ptr, col_ptr, &slices_, &ndc_,
                  &offsets_);
    cpumultiplyDpermuteMatrix(number_of_tiles, 1, ndc_, slices_, row_ptr, col_ptr,
                              val_ptr, &row_ptr, &col_ptr, &val_ptr, true);

    row_ptr_len = m_rows + 1;
    col_ptr_len = row_ptr[m_rows];
    tile_bound_len = number_of_tiles + 1;
    intra_tile_sep_len = number_of_tiles;

    hipMalloc((void**)&d_row_ptr, row_ptr_len * sizeof(int));
    hipMalloc((void**)&d_col_ptr, col_ptr_len * sizeof(int));
    hipMalloc((void**)&d_tile_boundaries, tile_bound_len * sizeof(int));
    hipMalloc((void**)&d_intra_tile_sep, intra_tile_sep_len * sizeof(int));

    hipMemcpy(d_row_ptr, row_ptr, row_ptr_len * sizeof(int),
              hipMemcpyHostToDevice);
    hipMemcpy(d_col_ptr, col_ptr, col_ptr_len * sizeof(int),
              hipMemcpyHostToDevice);
    hipMemcpy(d_tile_boundaries, ndc_, tile_bound_len * sizeof(int),
              hipMemcpyHostToDevice);
    hipMemcpy(d_intra_tile_sep, offsets_, intra_tile_sep_len * sizeof(int),
              hipMemcpyHostToDevice);
  }

  // virtual void TearDown() will be called after each test is run.
  // You should define it if there is cleanup work to do.  Otherwise,
  // you don't have to provide it.
  //
  void TearDown() override {
    hipFree(d_row_ptr);
    hipFree(d_col_ptr);
    hipFree(d_tile_boundaries);
    hipFree(d_intra_tile_sep);
    delete ndc_;
    delete slices_;
    delete offsets_;

    delete row_ptr;
    delete col_ptr;
    delete val_ptr;
  }

  // Declares the variables your tests want to use.
  uint number_of_tiles;
  int m_rows;

  int* row_ptr;
  int* col_ptr;
  double* val_ptr;  // create pointers for matrix in csr format

  int* ndc_;     // array with indices of each tile in all slices
  int* slices_;  // array with nodes grouped in slices
  int* offsets_;

  int* d_row_ptr;
  int* d_col_ptr;
  int* d_tile_boundaries;
  int* d_intra_tile_sep;
  size_t row_ptr_len;
  size_t col_ptr_len;
  size_t tile_bound_len;
  size_t intra_tile_sep_len;
};
// When you have a test fixture, you define a test using TEST_F
// instead of TEST.

TEST_F(ColoringEnv, FillShmem) {

  uint max_nodes, max_edges;
  get_MaxTileSize(number_of_tiles, ndc_, row_ptr, &max_nodes, &max_edges);
    // calc shMem
  size_t shMem_bytes = (max_nodes+1 + max_edges) * sizeof(int);
  dim3 gridSize(number_of_tiles);
  dim3 blockSize(512);

  bool* errors;
  hipMallocManaged((void**)&errors, sizeof(bool));
  *errors = false;

  ShMemLoadTest<<<gridSize, blockSize, shMem_bytes>>>(
      d_row_ptr, d_col_ptr, d_tile_boundaries,
      max_nodes, max_edges, errors);
  hipDeviceSynchronize();

  EXPECT_FALSE(*errors);
}

// Tests the default c'tor.
TEST_F(ColoringEnv, CountersReduction) {
  using namespace red_test;
  // You can access data in the test fixture here.

  uint num_Blocks = 2;
  uint num_Counters = 50;
  std::vector<Counters> a(num_Counters);
  for (size_t i = 0; i < num_Counters; i++){
    for (size_t m_idx = 0; m_idx < max_bit_width; m_idx++)
    {
      a[i].m[m_idx] = i*max_bit_width + m_idx;
    }
  }
  Counters sum_c = std::reduce(a.begin(), a.end(), Counters{}, Sum_Counters());
  Counters max_c = std::reduce(a.begin(), a.end(), Counters{}, Max_Counters());

  Counters* d_counters_in;
  Counters* d_counters_out;
  hipMalloc((void**)&d_counters_in, num_Counters * sizeof(Counters));
  hipMalloc((void**)&d_counters_out, 2 * num_Blocks * sizeof(Counters));
  hipMemcpy(d_counters_in, a.data(), num_Counters * sizeof(Counters),
            hipMemcpyHostToDevice);

  dim3 gridSize(num_Blocks);
  dim3 blockSize(25);


  SingleReduce<<<gridSize, blockSize>>>(d_counters_in, num_Counters, d_counters_out);
  hipDeviceSynchronize();

  Counters d_SumResult;
  Counters d_MaxResult;
  hipMemcpy(&d_SumResult, d_counters_out, sizeof(Counters),
            hipMemcpyDeviceToHost);
  hipMemcpy(&d_MaxResult, d_counters_out + 1, sizeof(Counters),
            hipMemcpyDeviceToHost);

  for (size_t i = 0; i < max_bit_width; i++)
  {
    EXPECT_EQ(sum_c.m[i], d_SumResult.m[i]);
    EXPECT_EQ(max_c.m[i], d_MaxResult.m[i]);
  }
}


TEST_F(ColoringEnv, CPU_Comparison) {
  uint max_nodes, max_edges;
  get_MaxTileSize(number_of_tiles, ndc_, row_ptr, &max_nodes, &max_edges);
  
  // calc shMem
  size_t shMem_bytes = (max_nodes+1 + max_edges) * sizeof(int);
  dim3 gridSize(number_of_tiles);
  dim3 blockSize(512);

  Counters* d_results;
  hipMalloc((void**)&d_results, number_of_tiles * 2 *sizeof(Counters));

  // run GPU version
  coloring1Kernel<<<gridSize, blockSize, shMem_bytes>>>(
      d_row_ptr, d_col_ptr, d_tile_boundaries, d_intra_tile_sep, m_rows,
      max_nodes, max_edges, d_results);
  hipDeviceSynchronize();

  Counters gpu_total;
  hipMemcpy(&gpu_total, d_results, 1 * sizeof(Counters),
            hipMemcpyDeviceToHost);
  Counters gpu_max;
  hipMemcpy(&gpu_max, d_results + 1, 1 * sizeof(Counters),
            hipMemcpyDeviceToHost);


  Counters cpu_max, cpu_total;
  cpu_dist1(row_ptr, col_ptr, m_rows, &cpu_total, &cpu_max);

  for (int i = 0; i < max_bit_width; ++i){
    EXPECT_EQ(cpu_total.m[i], gpu_total.m[i]);
    EXPECT_EQ(cpu_max.m[i], gpu_max.m[i]);
  }
  

  hipFree(d_results);
}

}  // namespace